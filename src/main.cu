#include "hip/hip_runtime.h"
#include <iostream>
#include "data.h"
#include "cuda_helpers.cuh"
#include "GPUTree.cuh"

void printTreeRecurse(std::vector< TreeNode> &tree, int index, int level, data&d){

	if (index >= tree.size()){
		return;
	}

	for (int i = 0; i < level; i++){
		std::cout << "\t";
	}

	if (tree[index].infogain == 0){
		std::cout << "<-->\n";

	}
	else{
		std::cout << "<" << d.attribute_names[tree[index].attributeIndex] << ", " << tree[index].attributeValue << ", " << tree[index].infogain << ", (";
		printf("%1.2f,%1.2f)>\n", tree[index].left_prob, tree[index].right_prob);
	}

	printTreeRecurse(tree, index * 2 + 1, level + 1,d);
	printTreeRecurse(tree, index * 2 + 2, level + 1,d);


}

void printTree(std::vector<TreeNode> &tree, data&d){
	std::cout << "<attribute, attribute value, infogain, (left_prob, right_prob)>\n";
	printTreeRecurse(tree, 0, 0,d);
}

int main(int argc, char **argv)
{

	if (argc != 3){
		std::cout << "usage: GPUTree.exe <filename.arff> <n levels>\n";
		return 0;
	}

	std::string filename(argv[1]);
	int n_levels = std::atoi(argv[2]);

	//Force cuda context initialisation
	safe_cuda(hipFree(0));

	data d(filename);

	int max_nodes = std::pow(2, n_levels) - 1;
	std::vector<TreeNode> tree(max_nodes);

	try{
		Timer t;
		generate_tree( d.attributes_compacted.data(), d.attributes.size(),d.classes.data(), d.attributes[0].size(), n_levels, tree.data());
		hipDeviceSynchronize();
		t.printElapsed("Tree build time");
	}
	catch (thrust::system_error &e){
		std::cerr << e.what() << "\n";

	}

	printTree(tree, d);

	return 0;
}
